// matrix_mult.cu

#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel for matrix multiplication
__global__ void matrixMultiply(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    const int N = 16; // Matrix size (N x N)
    const int numElements = N * N;

    // Allocate memory on the host
    float* h_A = new float[numElements];
    float* h_B = new float[numElements];
    float* h_C = new float[numElements];

    // Initialize matrices h_A and h_B (fill with appropriate values)

    // Allocate memory on the device
    float* d_A, *d_B, *d_C;
    hipMalloc(&d_A, numElements * sizeof(float));
    hipMalloc(&d_B, numElements * sizeof(float));
    hipMalloc(&d_C, numElements * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_A, h_A, numElements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, numElements * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);

    // Launch the kernel
    matrixMultiply<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, numElements * sizeof(float), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Print the result (h_C)

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}