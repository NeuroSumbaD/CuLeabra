#include "hip/hip_runtime.h"
// matrix_multiply.cu
#include <iostream>
#include <vector>
// CUDA runtime
// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime_api.h>

__global__ float elementwise_multiply(float* A, float* B, float* C, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        C[idx] = A[idx] * B[idx];
    }
}

void multiply_matrices(float* A, float* B, float* C, int size) {
    int threads_per_block = 256;
    int num_blocks = (size + threads_per_block - 1) / threads_per_block;
    elementwise_multiply<<<num_blocks, threads_per_block>>>(A, B, C, size);
    hipDeviceSynchronize();
}